#include <stdlib.h>
#include <fstream>
#include <thrust/sort.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "aes_common.h"


int main( int argc, char **argv )
{
    if ( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n number of bits for the key (128, 192, 256)\n" );
        printf( "-i input file to be encrypted\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }

    int n = read_int( argc, argv, "-n", 128 );
    int num_rounds;
    if (n == 128) num_rounds = 10;
    else if (n == 192) num_rounds = 12;
    else if (n == 256) num_rounds = 14;
    else {
        printf("Key size needs to be either 128, 192, or 256 bits.");
        return 1;
    }


    char *savename = read_string( argc, argv, "-o", NULL );
    char *inputname = read_string( argc, argv, "-i", NULL );
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    FILE *finput = inputname ? fopen( inputname, "r" ) : NULL;
    if (finput == NULL) 
    {
        printf("Needs an input file.\n");
        return 1;
    }

    fseek(finput, 0L, SEEK_END);
    long filesize = ftell(finput);
    rewind(finput);
    fseek(finput, 0L, SEEK_SET);

    char* buf = (char*) malloc(sizeof(char) * filesize + 1);

    size_t read = fread(buf, sizeof(char), filesize, finput);
    if (read != filesize) 
    {
        printf("Read number of bytes was different than actual size.\n");
    }

    printf("input: \n%s\n", buf);

    double simulation_time = read_timer();
    // do the actual encryption


    simulation_time = read_timer() - simulation_time;
    // do decryption to verify correctness

    printf( "n = %d, simulation_time = %g seconds\n", n, simulation_time);

    if( fsave )
        fclose( fsave );

    return 0;
}

